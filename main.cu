
#include "hip/hip_runtime.h"


#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <float.h>

__global__  void vector_sub(double *out, const double *a, double c, int m, int n) {
    int tid =threadIdx.x;

    if (tid >= m && tid < n) {
        out[tid] -= a[tid] * c;
    }
}

void printMatrix(double **matrix, const int *SIZE) {
    for (int i = 0; i < *SIZE; ++i) {
        for (int j = 0; j < *SIZE; ++j) {
            printf("%f ", matrix[i][j]);
        }
        printf("\n");
    }
    printf("\n");
}

double diagonalMultiplication(double **matrix, const int *SIZE) {
    double rez = 1;
    for (int i = 0; i < *SIZE; ++i) rez *= matrix[i][i];
    return rez;
}

int zeroesCheck(const double *range, const int *SIZE) {
    int count = 0, flag = 1;
    for (int i = 0; i < *SIZE; ++i)
        if (range[i] == 0 && flag) {
            count++;
        } else flag = 0;
    return count;
}

int power(int a, int b) {
    int rez = 1;
    for (int i = 0; i < b; ++i) rez *= a;
    return rez;
}

int sort(double **matrix, int *SIZE) {
    int i, j, count = 0;
    double *temp;
    for (i = 0; i < *SIZE - 1; i++)
        for (j = 0; j < *SIZE - i - 1; j++) {
            if (zeroesCheck(matrix[j], SIZE) > zeroesCheck(matrix[j + 1], SIZE)) {
                count++;
                temp = matrix[j];
                matrix[j] = matrix[j + 1];
                matrix[j + 1] = temp;
            }
        }
    return power(-1, count);
}


double gaussianDeterminant(double **matrix, int* SIZE) {
    int size = *SIZE;
    double first, factor;
    double *d_a, *d_out;

    hipMalloc((void **) &d_a, sizeof(double) * *SIZE);
    hipMalloc((void **) &d_out, sizeof(double) * *SIZE);

    while (size > 1) {
        if (matrix[*SIZE - size][*SIZE - size] == 0) return 0;
        first = matrix[*SIZE - size][*SIZE - size];
        for (int i = *SIZE - size + 1; i < *SIZE; ++i) {

            factor = matrix[i][*SIZE - size] / first;
            hipMemcpy(d_out, matrix[i], sizeof(double) * *SIZE, hipMemcpyHostToDevice);
            hipMemcpy(d_a, matrix[*SIZE - size], sizeof(double) * *SIZE, hipMemcpyHostToDevice);

            vector_sub <<< 1, *SIZE >>>(d_out, d_a, factor, *SIZE - size, *SIZE);
            hipMemcpy(matrix[i], d_out, sizeof(double) * *SIZE, hipMemcpyDeviceToHost);
            hipDeviceSynchronize();
        }

        size--;
    }
    hipFree(d_a);
    hipFree(d_out);
    return diagonalMultiplication(matrix, SIZE);
}

void init() {
    FILE *fp1, *fp2;
    if ((fp1 = fopen("read.txt", "r")) == nullptr) {
        printf("Can't open file 'read.txt'\n");
        exit(-1);
    }
    if ((fp2 = fopen("write.txt", "w")) == nullptr) {
        printf("Can't open file 'write.txt'\n");
        exit(-1);
    }
    double **matrix;
    double determinant;
    int SIZE, sign;
    clock_t time_start, time_finish;

    while (fscanf(fp1, "%d", &SIZE) == 1) {
        matrix = (double **) malloc(SIZE * sizeof(double *));
        for (int i = 0; i < SIZE; ++i) {
            matrix[i] = (double *) malloc(SIZE * sizeof(double));
            for (int j = 0; j < SIZE; ++j) {
                fscanf(fp1, "%lf", &matrix[i][j]);
            }
        }
        time_start = clock();
        sign = sort(matrix, &SIZE);
        determinant = gaussianDeterminant(matrix, &SIZE) * (double) sign;
        time_finish = clock();
        fprintf(fp2, "%ld %f\n", time_finish - time_start, determinant);
        for (int i = 0; i < SIZE; ++i) free(matrix[i]);
        free(matrix);
        if (determinant > DBL_MAX) exit(-2);
    }
    fclose(fp1);
    fclose(fp2);
}

int main() {
    init();
    return 0;
}




